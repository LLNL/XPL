#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "xpl-tracer.h"

__global__ void clearImageSequence(int *root)
{
  const int x = (int )(blockIdx . x * blockDim . x * blockDim . y + threadIdx . y * blockDim . x + threadIdx . x);
  traceW( *(root + x)) = 0;
}
int img[40960];

int main()
{
  clearImageSequence<<<1,1>>>(img);
  hipDeviceSynchronize();
  return 0;
}

template<> inline __gnu_cxx::__enable_if< true,void > ::__type std::__fill_a(_Bit_type *__first,_Bit_type *__last,const int &__value)
{
  const int __tmp = traceR(__value);
  for (; __first != __last; traceR(++__first)) 
    traceW( *__first) = ((_Bit_type )__tmp);
}

template<> inline void std::fill(_Bit_type *__first,_Bit_type *__last,const int &__value)
{
  ;
  __fill_a((__niter_base(__first)),(__niter_base(__last)),(traceR(__value)));
}
