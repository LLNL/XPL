#include <hip/hip_runtime.h>
#include "xpl-tracer.h"

void setPtrVal(int *x,int val)
{
  traceW( *x) = val;
}

int main()
{
  int xyz[100];
  
#pragma xpl diagnostic tracerPrint(std::cerr)
  tracerPrint(std::cerr);
  setPtrVal(xyz + 1,0);
  
#pragma xpl diagnostic tracerPrint(std::cout)
  tracerPrint(std::cout);
  return 0;
}

template<> inline __gnu_cxx::__enable_if< true,void > ::__type std::__fill_a(_Bit_type *__first,_Bit_type *__last,const int &__value)
{
  const int __tmp = traceR(__value);
  for (; __first != __last; traceR(++__first)) 
    traceW( *__first) = ((_Bit_type )__tmp);
}

template<> inline void std::fill(_Bit_type *__first,_Bit_type *__last,const int &__value)
{
  ;
  __fill_a((__niter_base(__first)),(__niter_base(__last)),(traceR(__value)));
}
